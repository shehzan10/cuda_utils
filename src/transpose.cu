#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""

//NVTX Dir: C:\Program Files\NVIDIA GPU Computing Toolkit\nvToolsExt
#include <nvToolsExt.h>

//Initialize sizes
const int rows = 4096;
const int cols = 4096;
const int BLOCK_SIZE_X = 32;
const int BLOCK_SIZE_Y = 32;

//For unrolled transpose
const int TILE = 32;
const int SIDE = 8;

using namespace std;

#define NAIVE_TRANSPOSE      1
#define SHARED_MEM_TRANSPOSE 1
#define BANK_CONF_TRANSPOSE  1
#define UNROLLED_TRANSPOSE   1

struct DIMS
{
    dim3 dimBlock;
    dim3 dimGrid;
};

#define CUDA(call) do {                             \
    hipError_t e = (call);                         \
    if (e == hipSuccess) break;                    \
    fprintf(stderr, __FILE__":%d: %s (%d)\n",       \
            __LINE__, hipGetErrorString(e), e);    \
    exit(1);                                        \
} while (0)

double diffclock( clock_t clock1, clock_t clock2 )
{
    double diffticks = clock1 - clock2;
    double diffms    = diffticks / ( CLOCKS_PER_SEC / 1000.0);
    return diffms;
}

inline unsigned divup(unsigned n, unsigned div)
{
    return (n + div - 1) / div;
}

// Check errors
void postprocess(const float *ref, const float *res, int n)
{
    bool passed = true;
    for (int i = 0; i < n; i++)
    {
        if (res[i] != ref[i])
        {
            printf("ID:%d \t Res:%f \t Ref:%f\n", i, res[i], ref[i]);
            printf("%25s\n", "*** FAILED ***");
            passed = false;
            break;
        }
    }
    if(passed)
        printf("Post process check passed!!\n");
}

void preprocess(float *res, float *dev_res, int n)
{
    for (int i = 0; i < n; i++)
    {
        res[i] = -1;
    }
    hipMemset(dev_res, -1, n * sizeof(float));
}

__global__ void copyKernel(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // row
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // col

    int index_in = i*cols+j;   // (i,j) from matrix A

    b[index_in] = a[index_in];
}

__global__ void matrixTransposeNaive(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    //HINT: Look at copyKernel above

    int i = blockIdx.y * blockDim.y + threadIdx.y;  // row
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // col

    int index_in  =  i*cols+j;      // Compute input index (i,j) from matrix A
    int index_out =  j*rows+i;      // Compute output index (j,i) in matrix B = transpose(A)

    // Copy data from A to B
    b[index_out] = a[index_in];
}

__global__ void matrixTransposeShared(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    //Allocate appropriate shared memory
    __shared__ float mat[BLOCK_SIZE_X][BLOCK_SIZE_Y];

    //Compute input and output index
    int bx = blockIdx.x * BLOCK_SIZE_X;
    int by = blockIdx.y * BLOCK_SIZE_Y;
    int i  = by + threadIdx.y;        // row
    int j  = bx + threadIdx.x;        // col
    int ti = bx + threadIdx.y;        // row
    int tj = by + threadIdx.x;        // col

    //Copy data from input to shared memory
    if(i < rows && j < cols)
            mat[threadIdx.x][threadIdx.y] = a[i * cols + j];

    __syncthreads();

    //Copy data from shared memory to global memory
    if(tj < cols && ti < rows)
            b[ti * rows + tj] = mat[threadIdx.y][threadIdx.x];
}

__global__ void matrixTransposeSharedwBC(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    //HINT: Copy code from matrixTransposeShared kernel, while solving bank conflict problem
    __shared__ float mat[BLOCK_SIZE_X][BLOCK_SIZE_Y + 1];

    //Allocate appropriate shared memory

    //Compute input and output index
    int bx = blockIdx.x * BLOCK_SIZE_X;
    int by = blockIdx.y * BLOCK_SIZE_Y;
    int i  = by + threadIdx.y;        // row
    int j  = bx + threadIdx.x;        // col
    int ti = bx + threadIdx.y;        // row
    int tj = by + threadIdx.x;        // col

    //Copy data from input to shared memory
    if(i < rows && j < cols)
            mat[threadIdx.x][threadIdx.y] = a[i * cols + j];

    __syncthreads();

    //Copy data from shared memory to global memory
    if(tj < cols && ti < rows)
            b[ti * rows + tj] = mat[threadIdx.y][threadIdx.x];
}

__global__ void matrixTransposeUnrolled(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    //Allocate appropriate shared memory
    __shared__ float mat[TILE][TILE + 1];

    //Compute input and output index
    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;

    //Copy data from input to shared memory. Multiple copies per thread.
    #pragma unroll
    for(int k = 0; k < TILE ; k += SIDE)
    {
            if(x < rows && y + k < cols)
                    mat[threadIdx.y + k][threadIdx.x] = a[((y + k) * rows) + x];
    }

    __syncthreads();

    x = blockIdx.y * TILE + threadIdx.x;
    y = blockIdx.x * TILE + threadIdx.y;

    //Copy data from shared memory to global memory. Multiple copies per thread.
    #pragma unroll
    for(int k = 0; k < TILE; k += SIDE)
    {
            if(x < cols && y + k < rows)
                    b[(y + k) * cols + x] = mat[threadIdx.x][threadIdx.y + k];
    }
}

int main(int argc, char *argv[])
{
    //Run Memcpy benchmarks
    nvtxRangeId_t cudaBenchmark = nvtxRangeStart("CUDA Memcpy Benchmark");
#if defined WIN64
    system(".\\..\\bin\\cudaBenchmark.exe");
#elif defined LINUX
    system("./bin/cudaBenchmark");
#endif
    nvtxRangeEnd(cudaBenchmark);

    // Host arrays.
    float* a = new float[rows*cols];
    float* b = new float[rows*cols];
    float* a_gold = new float[rows*cols];
    float* b_gold = new float[rows*cols];

    // Device arrays
    float *d_a, *d_b;

    // Allocate memory on the device
    CUDA( hipMalloc((void **) &d_a, rows*cols*sizeof(float)) );

    CUDA( hipMalloc((void **) &d_b, rows*cols*sizeof(float)) );

    // Fill matrix A
    for (int i = 0; i < rows * cols; i++)
        a[i] = (float)i;

    cout << endl;

    // Copy array contents of A from the host (CPU) to the device (GPU)
    hipMemcpy(d_a, a, rows*cols*sizeof(float), hipMemcpyHostToDevice);

    //Compute "gold" reference standard
    for(int ii = 0; ii < rows; ii++)
    {
        for(int jj = 0; jj < cols; jj++)
        {
            a_gold[jj * rows + ii] = a[jj * cols + ii];
            b_gold[ii * cols + jj] = a[jj * cols + ii];
        }
    }

    hipDeviceSynchronize();

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    cout << "***Launch the transpose!***" << endl << endl;

#define CPU_TRANSPOSE
#ifdef CPU_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***CPU Transpose***" << endl;
    {
        // start the timer
        nvtxRangeId_t cpuBenchmark = nvtxRangeStart("CPU Transpose Benchmark");

        clock_t begin = clock();
        int iters = 10;
        for (int k=0; k<iters; k++)
        {
            for(int ii = 0; ii < rows; ii++)
                for(int jj = 0; jj < cols; jj++)
                    b[ii * cols + jj] = a[jj * cols + ii];
        }
        // stop the timer
        clock_t end = clock();
        nvtxRangeEnd(cpuBenchmark);

        float time = 0.0f;
        time = diffclock(end, begin);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(rows*cols*sizeof(float)) / (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Device To Device Copy***" << endl;
    {
        preprocess(b, d_b, rows*cols);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"

        DIMS dims;
        dims.dimBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
        dims.dimGrid  = dim3(divup(rows, BLOCK_SIZE_X),
                             divup(cols, BLOCK_SIZE_Y),
                             1
                             );

        // start the timer
        nvtxRangeId_t naiveBenchmark = nvtxRangeStart("Device to Device Copy");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            copyKernel<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(naiveBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(rows*cols*sizeof(float)) /
                            (1000.0*1000.0*1000.0*time);        //2.0 for read of A and read and write of B
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        hipMemcpy(b, d_b, cols*rows*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(a_gold, b, rows * cols);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////

#if NAIVE_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Naive Transpose***" << endl;
    {
        preprocess(b, d_b, rows*cols);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        // HINT: Look above for copy kernel dims computation
        DIMS dims;
        dims.dimBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
        dims.dimGrid  = dim3(divup(rows, BLOCK_SIZE_X),
                             divup(cols, BLOCK_SIZE_Y),
                             1);

        // start the timer
        nvtxRangeId_t naiveBenchmark = nvtxRangeStart("Naive Transpose Benchmark");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            matrixTransposeNaive<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(naiveBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(rows*cols*sizeof(float)) /
            (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        hipMemcpy(b, d_b, cols*rows*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, rows * cols);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

#if SHARED_MEM_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Shared Memory Transpose***" << endl;
    {
        preprocess(b, d_b, rows*cols);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        DIMS dims;
        dims.dimBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
        dims.dimGrid  = dim3(divup(rows, BLOCK_SIZE_X),
                             divup(cols, BLOCK_SIZE_Y),
                             1);

        // start the timer
        nvtxRangeId_t sharedMemBenchmark = nvtxRangeStart("Shared Memory Transpose Benchmark");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            matrixTransposeShared<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(sharedMemBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(rows*cols*sizeof(float)) /
            (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        hipMemcpy(b, d_b, cols*rows*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, rows * cols);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

#if BANK_CONF_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Without Bank Conflicts Transpose***" << endl;
    {
        preprocess(b, d_b, rows*cols);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        DIMS dims;
        dims.dimBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
        dims.dimGrid  = dim3(divup(rows, BLOCK_SIZE_X),
                             divup(cols, BLOCK_SIZE_Y),
                             1);

        // start the timer
        nvtxRangeId_t sharedMemBenchmark = nvtxRangeStart("Shared Memory Transpose Benchmark");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            matrixTransposeSharedwBC<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(sharedMemBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(rows*cols*sizeof(float)) /
            (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        hipMemcpy(b, d_b, cols*rows*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, rows * cols);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

#if UNROLLED_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Unrolled Loops Transpose***" << endl;
    {
        preprocess(b, d_b, rows*cols);
        // Assign a 2D distribution of TILE x SIDE x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        DIMS dims;
        dims.dimBlock = dim3(TILE, SIDE, 1);
        dims.dimGrid  = dim3(divup(rows, TILE),
                             divup(cols, TILE),
                             1);

        // start the timer
        nvtxRangeId_t unrolledBenchmark = nvtxRangeStart("Shared Memory Transpose Benchmark");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            matrixTransposeUnrolled<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(unrolledBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(rows*cols*sizeof(float)) /
            (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        hipMemcpy(b, d_b, cols*rows*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, rows * cols);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif
    // copy the answer back to the host (CPU) from the device (GPU)

    /*
       cout << "Entries of B: \n";
       for (int i = 0; i < 32; i++) {
       cout << b[i] << " ";
       }
       cout << endl;
       for (int i = 0; i < 32; i++) {
       cout << b[i * cols] << " ";
       }
       cout << endl;

     */

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    delete[] a;
    delete[] b;

    //Destroy Events
    CUDA(hipEventDestroy(start));
    CUDA(hipEventDestroy(stop));

    //CUDA Reset for NVProf
    CUDA(hipDeviceReset());

    // successful program termination
    return 0;
}
