#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""

//NVTX Dir: C:\Program Files\NVIDIA GPU Computing Toolkit\nvToolsExt
#include <nvToolsExt.h>

#include "memBenchmark.h"

//Initialize sizes
const int sizeX = 4096;
const int sizeY = 4096;
const int BLOCK_SIZE_X = 32;
const int BLOCK_SIZE_Y = 32;

//For unrolled transpose
const int TILE = 32;
const int SIDE = 8;

using namespace std;

#define NAIVE_TRANSPOSE      1
#define SHARED_MEM_TRANSPOSE 1
#define BANK_CONF_TRANSPOSE  1
#define UNROLLED_TRANSPOSE   1

struct DIMS
{
    dim3 dimBlock;
    dim3 dimGrid;
};

#define CUDA(call) do {                             \
    hipError_t e = (call);                         \
    if (e == hipSuccess) break;                    \
    fprintf(stderr, __FILE__":%d: %s (%d)\n",       \
            __LINE__, hipGetErrorString(e), e);    \
    exit(1);                                        \
} while (0)

double diffclock( clock_t clock1, clock_t clock2 )
{
    double diffticks = clock1 - clock2;
    double diffms    = diffticks / ( CLOCKS_PER_SEC / 1000.0);
    return diffms;
}

inline unsigned divup(unsigned n, unsigned div)
{
    return (n + div - 1) / div;
}

// Check errors
void postprocess(const float *ref, const float *res, int n)
{
    bool passed = true;
    for (int i = 0; i < n; i++)
    {
        if (res[i] != ref[i])
        {
            printf("ID:%d \t Res:%f \t Ref:%f\n", i, res[i], ref[i]);
            printf("%25s\n", "*** FAILED ***");
            passed = false;
            break;
        }
    }
    if(passed)
        printf("Post process check passed!!\n");
}

void preprocess(float *res, float *dev_res, int n)
{
    for (int i = 0; i < n; i++)
    {
        res[i] = -1;
    }
    hipMemset(dev_res, -1, n * sizeof(float));
}

__global__ void copyKernel(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  //
    int j = blockIdx.y * blockDim.y + threadIdx.y;  //

    int index_in = j * sizeX + i;   // (i,j) from matrix A

    b[index_in] = a[index_in];
}

__global__ void matrixTransposeNaive(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    //HINT: Look at copyKernel above

    int i = blockIdx.x * blockDim.x + threadIdx.x;  //
    int j = blockIdx.y * blockDim.y + threadIdx.y;  //

    int index_in  = j * sizeX + i;      // Compute input index (i,j) from matrix A
    int index_out = i * sizeY + j;      // Compute output index (j,i) in matrix B = transpose(A)

    // Copy data from A to B
    b[index_out] = a[index_in];
}

__global__ void matrixTransposeShared(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    //Allocate appropriate shared memory
    __shared__ float mat[BLOCK_SIZE_Y][BLOCK_SIZE_X];

    //Compute input and output index
    int bx = blockIdx.x * BLOCK_SIZE_X;
    int by = blockIdx.y * BLOCK_SIZE_Y;
    int i  = bx + threadIdx.x;        //
    int j  = by + threadIdx.y;        //
    int ti = by + threadIdx.x;        //
    int tj = bx + threadIdx.y;        //

    //Copy data from input to shared memory
    if(i < sizeX && j < sizeY)
            mat[threadIdx.y][threadIdx.x] = a[j * sizeX + i];

    __syncthreads();

    //Copy data from shared memory to global memory
    if(ti < sizeY && tj < sizeX)
            b[tj * sizeY + ti] = mat[threadIdx.x][threadIdx.y];
}

__global__ void matrixTransposeSharedwBC(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    //HINT: Copy code from matrixTransposeShared kernel, while solving bank conflict problem
    //Allocate appropriate shared memory
    __shared__ float mat[BLOCK_SIZE_Y][BLOCK_SIZE_X + 1];

    //Compute input and output index
    int bx = blockIdx.x * BLOCK_SIZE_X;
    int by = blockIdx.y * BLOCK_SIZE_Y;
    int i  = bx + threadIdx.x;        //
    int j  = by + threadIdx.y;        //
    int ti = by + threadIdx.x;        //
    int tj = bx + threadIdx.y;        //

    //Copy data from input to shared memory
    if(i < sizeX && j < sizeY)
            mat[threadIdx.y][threadIdx.x] = a[j * sizeX + i];

    __syncthreads();

    //Copy data from shared memory to global memory
    if(ti < sizeY && tj < sizeX)
            b[tj * sizeY + ti] = mat[threadIdx.x][threadIdx.y];
}

__global__ void matrixTransposeUnrolled(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    //Allocate appropriate shared memory
    __shared__ float mat[TILE][TILE + 1];

    //Compute input and output index
    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;

    //Copy data from input to shared memory. Multiple copies per thread.
    #pragma unroll
    for(int k = 0; k < TILE ; k += SIDE)
    {
            //if(x < sizeX && y + k < sizeY)
                    mat[threadIdx.y + k][threadIdx.x] = a[((y + k) * sizeX) + x];
    }

    __syncthreads();

    x = blockIdx.y * TILE + threadIdx.x;
    y = blockIdx.x * TILE + threadIdx.y;

    //Copy data from shared memory to global memory. Multiple copies per thread.
    #pragma unroll
    for(int k = 0; k < TILE; k += SIDE)
    {
            //if(x < sizeY && y + k < sizeX)
                    b[(y + k) * sizeY + x] = mat[threadIdx.x][threadIdx.y + k];
    }
}

__global__ void copyKernelUnrolled(const float* __restrict__ const a,
        float* __restrict__ const b)
{
    int i = blockIdx.x * TILE + threadIdx.x;  //
    int j = blockIdx.y * TILE + threadIdx.y;  //

    #pragma unroll
    for(int k = 0; k < TILE ; k += SIDE)
    {
        int index_in = (j + k) * sizeX + i;   // (i,j) from matrix A
        b[index_in] = a[index_in];
    }
}


int main(int argc, char *argv[])
{
    int runtime = 0;
    CUDA(hipRuntimeGetVersion(&runtime));
    printf("Runtime = %d\n", runtime);

    //Run Memcpy benchmarks
    nvtxRangeId_t cudaBenchmark = nvtxRangeStart("CUDA Memcpy Benchmark");
    memBenchmark();
    nvtxRangeEnd(cudaBenchmark);

    // Set mapping
    hipSetDeviceFlags(hipDeviceMapHost);

    // Host arrays.
    float* a      = NULL;// = new float[sizeX * sizeY];
    float* b      = NULL;// = new float[sizeX * sizeY];
    float* a_gold = NULL;// = new float[sizeX * sizeY];
    float* b_gold = NULL;// = new float[sizeX * sizeY];
    CUDA(hipHostAlloc((void **)&a,      sizeX * sizeY * sizeof(float), hipHostMallocMapped));
    CUDA(hipHostAlloc((void **)&b,      sizeX * sizeY * sizeof(float), hipHostMallocMapped));
    CUDA(hipHostAlloc((void **)&a_gold, sizeX * sizeY * sizeof(float), hipHostMallocMapped));
    CUDA(hipHostAlloc((void **)&b_gold, sizeX * sizeY * sizeof(float), hipHostMallocMapped));

    // Device arrays
    float *d_a, *d_b;

    // Allocate memory on the device
    //CUDA(hipMalloc((void **) &d_a, sizeX * sizeY * sizeof(float)));

    //CUDA(hipMalloc((void **) &d_b, sizeX * sizeY * sizeof(float)));
    CUDA(hipHostGetDevicePointer((void **)&d_a, (void *) a, 0));
    CUDA(hipHostGetDevicePointer((void **)&d_b, (void *) b, 0));

    // Fill matrix A
    for (int i = 0; i < sizeX * sizeY; i++)
        a[i] = (float)i;

    cout << endl;

    // Copy array contents of A from the host (CPU) to the device (GPU)
    //hipMemcpy(d_a, a, sizeX * sizeY * sizeof(float), hipMemcpyHostToDevice);

    //Compute "gold" reference standard
    for(int jj = 0; jj < sizeY; jj++)
    {
        for(int ii = 0; ii < sizeX; ii++)
        {
            a_gold[jj * sizeX + ii] = a[jj * sizeX + ii];
            b_gold[ii * sizeY + jj] = a[jj * sizeX + ii];
        }
    }

    hipDeviceSynchronize();

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    cout << "***Launch the transpose!***" << endl << endl;

#define CPU_TRANSPOSE
#ifdef CPU_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***CPU Transpose***" << endl;
    {
        // start the timer
        nvtxRangeId_t cpuBenchmark = nvtxRangeStart("CPU Transpose Benchmark");

        clock_t begin = clock();
        int iters = 10;
        for (int k=0; k<iters; k++)
        {
            for(int jj = 0; jj < sizeY; jj++)
                for(int ii = 0; ii < sizeX; ii++)
                    b[ii * sizeX + jj] = a[jj * sizeX + ii];
        }
        // stop the timer
        clock_t end = clock();
        nvtxRangeEnd(cpuBenchmark);

        float time = 0.0f;
        time = diffclock(end, begin);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(sizeX * sizeY*sizeof(float)) / (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Device To Device Copy***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"

        DIMS dims;
        dims.dimBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
        dims.dimGrid  = dim3(divup(sizeX, BLOCK_SIZE_X),
                             divup(sizeY, BLOCK_SIZE_Y),
                             1);

        // start the timer
        nvtxRangeId_t naiveBenchmark = nvtxRangeStart("Device to Device Copy");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            copyKernel<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(naiveBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(sizeX * sizeY*sizeof(float)) /
                            (1000.0*1000.0*1000.0*time);        //2.0 for read of A and read and write of B
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        //hipMemcpy(b, d_b, sizeY*sizeX*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(a_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////

#if NAIVE_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Naive Transpose***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        // HINT: Look above for copy kernel dims computation
        DIMS dims;
        dims.dimBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
        dims.dimGrid  = dim3(divup(sizeX, BLOCK_SIZE_X),
                             divup(sizeY, BLOCK_SIZE_Y),
                             1);

        // start the timer
        nvtxRangeId_t naiveBenchmark = nvtxRangeStart("Naive Transpose Benchmark");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            matrixTransposeNaive<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(naiveBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(sizeX * sizeY*sizeof(float)) /
            (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        //hipMemcpy(b, d_b, sizeY*sizeX*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

#if SHARED_MEM_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Shared Memory Transpose***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        DIMS dims;
        dims.dimBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
        dims.dimGrid  = dim3(divup(sizeX, BLOCK_SIZE_X),
                             divup(sizeY, BLOCK_SIZE_Y),
                             1);

        // start the timer
        nvtxRangeId_t sharedMemBenchmark = nvtxRangeStart("Shared Memory Transpose Benchmark");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            matrixTransposeShared<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(sharedMemBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(sizeX * sizeY*sizeof(float)) /
            (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        //hipMemcpy(b, d_b, sizeY*sizeX*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

#if BANK_CONF_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Without Bank Conflicts Transpose***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        DIMS dims;
        dims.dimBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
        dims.dimGrid  = dim3(divup(sizeX, BLOCK_SIZE_X),
                             divup(sizeY, BLOCK_SIZE_Y),
                             1);

        // start the timer
        nvtxRangeId_t sharedMemBenchmark = nvtxRangeStart("Shared Memory Transpose Benchmark");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            matrixTransposeSharedwBC<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(sharedMemBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(sizeX * sizeY*sizeof(float)) /
            (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        //hipMemcpy(b, d_b, sizeY*sizeX*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

#if UNROLLED_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Unrolled Loops Transpose***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of TILE x SIDE x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        DIMS dims;
        dims.dimBlock = dim3(TILE, SIDE, 1);
        dims.dimGrid  = dim3(divup(sizeX, TILE),
                             divup(sizeY, TILE),
                             1);

        // start the timer
        nvtxRangeId_t unrolledBenchmark = nvtxRangeStart("Shared Memory Transpose Benchmark");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            matrixTransposeUnrolled<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(unrolledBenchmark);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(sizeX * sizeY*sizeof(float)) /
            (1000.0*1000.0*1000.0*time);
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        //hipMemcpy(b, d_b, sizeY*sizeX*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Device To Device Copy Unrolled***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of TILE x SIDE x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        DIMS dims;
        dims.dimBlock = dim3(TILE, SIDE, 1);
        dims.dimGrid  = dim3(divup(sizeX, TILE),
                             divup(sizeY, TILE),
                             1);

        // start the timer
        nvtxRangeId_t copyBenchmarkUnrolled = nvtxRangeStart("Device to Device Copy Unrolled");
        hipEventRecord( start, 0);

        int iters = 10;
        for (int i=0; i<iters; i++)
        {
            // Launch the GPU kernel
            copyKernelUnrolled<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);
        }
        // stop the timer
        hipEventRecord( stop, 0);
        hipEventSynchronize( stop );
        nvtxRangeEnd(copyBenchmarkUnrolled);

        float time = 0.0f;
        hipEventElapsedTime( &time, start, stop);

        // print out the time required for the kernel to finish the transpose operation
        double Bandwidth = (double)iters*2.0*1000.0*(double)(sizeX * sizeY*sizeof(float)) /
                            (1000.0*1000.0*1000.0*time);        //2.0 for read of A and read and write of B
        cout << "Elapsed Time for " << iters << " runs = " << time << "ms" << endl;
        cout << "Bandwidth (GB/s) = " << Bandwidth << endl;

        // copy the answer back to the host (CPU) from the device (GPU)
        //hipMemcpy(b, d_b, sizeY*sizeX*sizeof(float), hipMemcpyDeviceToHost);

        postprocess(a_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////

    // copy the answer back to the host (CPU) from the device (GPU)

    /*
       cout << "Entries of B: \n";
       for (int i = 0; i < 32; i++) {
       cout << b[i] << " ";
       }
       cout << endl;
       for (int i = 0; i < 32; i++) {
       cout << b[i * sizeY] << " ";
       }
       cout << endl;

     */

    // free device memory
    //CUDA(hipFree(d_a));
    //CUDA(hipFree(d_b));

    // free host memory
    //delete[] a;
    //delete[] b;
    CUDA(hipHostFree(a));
    CUDA(hipHostFree(b));
    CUDA(hipHostFree(a_gold));
    CUDA(hipHostFree(b_gold));

    //Destroy Events
    CUDA(hipEventDestroy(start));
    CUDA(hipEventDestroy(stop));

    //CUDA Reset for NVProf
    CUDA(hipDeviceReset());

    // successful program termination
    return 0;
}
